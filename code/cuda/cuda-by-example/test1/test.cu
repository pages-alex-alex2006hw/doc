#include "hip/hip_runtime.h"
#include <iostream>
#include "../common/book.h"

__global__ void add(int a, int b, int *c)
{
}

int main(void) 
{
  // c allocated on stack
  int c;

  // hipMalloc allocates memory on the device
  // arg1: pointer to pointer to a memory
  //       &dev_c :-----> dev_c :-----> MEM
  //                      
  // we are creating a place holder for the argument!
  int *dev_c;
  HANDLE_ERROR(hipMalloc((void **) &dev_c, sizeof(int)));
  
  add<<<1,1>>>(2, 7, dev_c);

  HANDLE_ERROR(hipMemcpy(&c,
                          dev_c,
                          sizeof(int),
                          hipMemcpyDeviceToHost));
  printf("2 + 7 = %d\n", c);
  hipFree(dev_c);

  return 0;
}


