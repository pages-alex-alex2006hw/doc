#include "hip/hip_runtime.h"
/**
 * @file tcam.cu
 * @brief CUDA implementation of TCAM
 *
 * @author cjeong
 */
#include <hip/hip_runtime.h>
#include "tcam.h"

__global__ void tcam_kernel(unsigned *dev_memx, unsigned *dev_memy, unsigned a, 
                            unsigned *dev_d, unsigned *dev_be, unsigned fe, 
                            unsigned bse, unsigned clk, unsigned read, 
                            unsigned write, unsigned search, unsigned colwrite,
                            unsigned bitpos, unsigned *dev_q, unsigned *dev_mll)
{
  __shared__ int cache[CUDA_NTHREADS];
  int tid = threadIdx.x + blockIdx.x + blockDim.x;
  
  /* STEP #1: Initialize the cache entries to 0. */
  cache[threadIdx.x] = 0;
  __syncthreads();
  
  /* STEP #2: Each thread is responsible for checking 32-bit word of 
     a field in a TCAM word. In this step, we check if these 32-bit
     of the memory word matches the search data. */
  unsigned thr_bse = bse & (1 << (blockDim.x / TCAM_NBANKWORDS));
  if (tid < CUDA_N && thr_bse && clk) {
    unsigned thr_d =  dev_d[threadIdx.x];
    unsigned thr_be = dev_be[threadIdx.x];
    unsigned thr_fe = fe & (1 << (threadIdx.x / TCAM_NFIELDS));
    unsigned thr_memx = (tid < CUDA_N) ? dev_memx[tid] : (unsigned) -1;
    unsigned thr_memy = (tid < CUDA_N) ? dev_memy[tid] : (unsigned) -1;

    cache[threadIdx.x] = 
      ((~thr_fe | ~thr_be | (~thr_memx & ~thr_memy) |
        (thr_be & ~thr_d & thr_memx & ~thr_memy) |
        (thr_be & thr_d & ~thr_memx & thr_memy)) == (unsigned) -1);
  }
  __syncthreads();

  /* STEP #3: Perform reduction to determine MLL for this 32-bit word.
     After reduction, cache[0] will contain 1 iff MLL == 1 for this TCAM 
     word. */
  int cacheIdx = threadIdx.x;
  int i = blockDim.x/2;
  while (i != 0) {
    if (cacheIdx < i)
      cache[cacheIdx] = (cache[cacheIdx] == 1) && (cache[cacheIdx+i] == 1);
    __syncthreads();
    i = i/2;
  }

  /* STEP #4: the first thread in the thread block writes MLL result
     to global mem. */
  if (threadIdx.x == 0) 
    dev_mll[blockIdx.x] = cache[0];
}


void tcam(unsigned *dev_memx, unsigned *dev_memy, unsigned a, 
          unsigned *dev_d, unsigned *dev_be, unsigned fe, unsigned bse, 
          unsigned clk, unsigned read, unsigned write, unsigned search, 
          unsigned colwrite, unsigned bitpos, unsigned *dev_q, 
          unsigned *dev_mll)
{
  tcam_kernel<<<CUDA_NBLOCKS, CUDA_NTHREADS>>>(dev_memx, dev_memy, a, dev_d, 
                                               dev_be, fe, bse, clk, read, 
                                               write, search, colwrite, bitpos,
                                               dev_q, dev_mll);
}
